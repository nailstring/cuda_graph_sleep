#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

template <typename scalar_t>
__global__ void spin_kernel(
        torch::PackedTensorAccessor<scalar_t,1,torch::RestrictPtrTraits,size_t> sleep_time) {
  // see concurrentKernels CUDA sampl
  int64_t start_clock = clock64();
  int64_t clock_offset = 0;
  while (clock_offset < sleep_time[0])
  {
    clock_offset = clock64() - start_clock;
  }
}

void cuda_sleep(torch::Tensor sleep_time) {
    AT_DISPATCH_INTEGRAL_TYPES(sleep_time.type(), "sleep", ([&] {
        spin_kernel<scalar_t><<<1, 1, 0, at::cuda::getCurrentCUDAStream()>>>(
            sleep_time.packed_accessor<scalar_t,1,torch::RestrictPtrTraits,size_t>());
    }));
}
